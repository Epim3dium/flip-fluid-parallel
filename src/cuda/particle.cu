#include "hip/hip_runtime.h"
#include "particle.hpp"
#include "geometry_func.hpp"
#include "time.hpp"
#include <cassert>
#include <device_atomic_functions.h>
#include "vec2.hpp"
#include "AABB.hpp"
#include <SFML/Graphics/CircleShape.hpp>
#include <SFML/Graphics/RenderStates.hpp>
#include <array>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <execution>
#include <unordered_set>
#define CUDA_KERNEL_CHECK()                                                     \
do {                                                                            \
    hipError_t err = hipGetLastError();                                       \
    if (err != hipSuccess) {                                                   \
        fprintf(stderr, "Kernel launch error at %s:%d: %s\n",                   \
                __FILE__, __LINE__, hipGetErrorString(err));                   \
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
} while (0)
#define CUDA_CALL(call)                                                         \
do {                                                                            \
    hipError_t err = call;                                                     \
    if (err != hipSuccess) {                                                   \
        fprintf(stderr, "CUDA error in %s (%s:%d): %s\n",                       \
                #call, __FILE__, __LINE__, hipGetErrorString(err));            \
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
} while (0)

__global__ void accelerateKernel(Particles& particles, vec2f gravity, int max_count) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= max_count) return;
    particles.gpu_acceleration[i] += gravity;
}

void accelerate(Particles& particles, vec2f gravity) {
    int threadsPerBlock = 1024;
    int blocks = (max_particle_count + threadsPerBlock - 1) / threadsPerBlock;
    accelerateKernel<<<blocks, threadsPerBlock>>>(particles, gravity, max_particle_count);
    hipDeviceSynchronize();
    CUDA_KERNEL_CHECK();
}
__global__ void integrateKernel(Particles& particles, float dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= max_particle_count) return;
    particles.gpu_velocity[i] += particles.gpu_acceleration[i] * dt;
    particles.gpu_position[i] += particles.gpu_velocity[i] * dt;
    particles.gpu_acceleration[i] = {0, 0};
}
void integrate(Particles& particles, float dt) {
    int threadsPerBlock = 1024;
    int blocks = (max_particle_count + threadsPerBlock - 1) / threadsPerBlock;
    integrateKernel<<<blocks, threadsPerBlock>>>(particles, dt);
    hipDeviceSynchronize();
    CUDA_KERNEL_CHECK();
}
__device__ void resolveVelocities(vec2f* velocity, int p1, int p2, vec2f normal) {
    auto rel_vel = velocity[p1] - velocity[p2];
    auto rel_vel_normal = dot(rel_vel, normal);
    if (rel_vel_normal > 0) return;
    float restitution = 0.1f;
    float impulse = -(1 + restitution) * rel_vel_normal * 0.5;

    velocity[p1] += impulse * normal;
    velocity[p2] -= impulse * normal;

}
__device__ bool isColliding(const vec2f& v1, const vec2f& v2, float radius) {
    float2 d = make_float2(
        v1.x - v2.x,
        v1.y - v2.y
    );
    float distSquared = d.x * d.x + d.y * d.y;
    float combinedRadius = radius * 2.f;
    return distSquared < (combinedRadius * combinedRadius);
}
__device__ void processCollision(vec2f* position, vec2f* gpu_vels, int i, int ii, float radius) {
    auto diff = position[ii] - position[i];
    const float min_dist = radius * 2;
    if(isColliding(position[i], position[ii], radius)) {
        auto l = length(diff);
        auto n = diff / l;
        static const float damping = 0.2f;
        auto c = (min_dist - l) * 0.5f * damping;
        position[i] -= n * c;
        position[ii] += n * c;
        resolveVelocities(gpu_vels, i, ii, -n);
    }
}
void collide(Particles& particles) {
    assert(false);
}
struct CompactVec {
    uint32_t data[32];
    uint32_t size = 0;
};

__global__ void compareWithNeighbours(vec2f* position, vec2f* velocity, float radius, uint32_t* active, uint32_t* active_sizes, uint32_t checkerboard, int max_segs_cols, int max_segs_rows, const CompactVec* grid) {
    int max_size = max_segs_rows * max_segs_cols;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= active_sizes[checkerboard])
        return;
    auto row = active[i + max_size * checkerboard] / max_segs_cols;
    auto col = active[i + max_size * checkerboard] % max_segs_cols;
    auto& comp_vec1 = grid[row*max_segs_cols + col];
    if(comp_vec1.size == 0) 
        return;
    #define offset_grid(dirx, diry) &grid[(row+dirx) * max_segs_cols + col+diry]
    const CompactVec* comp_vecs[4] = {offset_grid(1, 0), offset_grid(0, 1), offset_grid(1, 1),offset_grid(1,-1)};

    for(int i = 0; i < comp_vec1.size; i++) {
        auto idx1 = comp_vec1.data[i];
        for(int ii = i + 1; ii < comp_vec1.size; ii++) {
            auto idx2 = comp_vec1.data[ii];
            processCollision(position, velocity, idx1, idx2, radius);
        }
        for(auto other : comp_vecs) {
            for(auto ii = 0; ii < other->size; ii++) {
                auto idx2 = (*other).data[ii];
                processCollision(position, velocity, idx1, idx2, radius);
            }
        }
    }
}
__global__ void clearGrid(
    CompactVec* col_grid,
    int col_grid_size) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= col_grid_size) return;
    col_grid[i].size = 0;
}
__global__ void assignParticlesToGrid(
    Particles& particles,
    CompactVec* col_grid,
    vec2f sim_area_min,
    int max_segs_cols,
    int max_segs_rows,
    uint32_t* active_flags, // Flattened grid of flags (per cell)
    uint32_t* active_flags_sizes
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= max_particle_count) return;

    vec2f pos = particles.gpu_position[i];
    uint32_t col = (pos.x - sim_area_min.x) / particles.diameter;
    uint32_t row = (pos.y - sim_area_min.y) / particles.diameter;

    if (col + 1 >= max_segs_cols || row + 1 >= max_segs_rows) return;

    int grid_idx = (row + 1) * max_segs_cols + (col + 1);
    CompactVec& comp_vec = col_grid[grid_idx];

    // Atomically get index to write into comp_vec
    uint32_t insert_idx = atomicAdd(&comp_vec.size, 1);
    if (insert_idx < 32U) {
        comp_vec.data[insert_idx] = i;
    }    
    bool old = (insert_idx != 0);  //only the first one insterts
    if(old == 1) return;

    int checkerboard = (row % 2)*2 + col%2;
    int max_size = max_segs_cols * max_segs_rows;
    auto idx = atomicAdd(&active_flags_sizes[checkerboard], 1);
    active_flags[idx+checkerboard*max_size] = grid_idx;
}
std::map<std::string, float> collide(Particles& particles, AABB sim_area) {
    std::map<std::string, float> result;
    const uint32_t max_segs_cols = sim_area.size().x / particles.diameter + 1 + 2;
    const uint32_t max_segs_rows = sim_area.size().y / particles.diameter + 1 + 2;

    static int col_grid_size = 0;
    static CompactVec* gpu_col_grid = nullptr;
    static uint32_t* gpu_active_idxs = nullptr;
    static uint32_t* gpu_active_sizes = nullptr;

    Stopwatch stop;
    if(col_grid_size != max_segs_rows * max_segs_cols) {
        if(col_grid_size != 0) {
            hipFree(gpu_col_grid);
            hipFree(gpu_active_idxs);
        }else {
            CUDA_CALL(hipMalloc(&gpu_active_sizes, sizeof(uint32_t) * 4U));
            CUDA_CALL(hipMemset(gpu_active_sizes, 0, sizeof(uint32_t) * 4U));
        }
        col_grid_size = max_segs_rows * max_segs_cols;
        CUDA_CALL(hipMalloc(&gpu_active_idxs, sizeof(uint32_t) * col_grid_size * 4U));
        CUDA_CALL(hipMalloc(&gpu_col_grid, sizeof(CompactVec) * col_grid_size));
        CUDA_CALL(hipMemset(gpu_col_grid, 0, sizeof(CompactVec) * col_grid_size));
    }
    result["particles::collide::allocate"] += stop.restart();


    int threadsPerBlock = 1024;
    int blocks = (max_particle_count + threadsPerBlock - 1) / threadsPerBlock;
    assignParticlesToGrid<<<blocks, threadsPerBlock>>>(
        particles, gpu_col_grid, 
        sim_area.min,
        max_segs_cols, max_segs_rows,
        gpu_active_idxs, gpu_active_sizes
    );
    hipDeviceSynchronize();
    result["particles::collide::assign"] += stop.restart();

    for(int i = 0; i < 4; i++) {
        compareWithNeighbours<<<(col_grid_size + threadsPerBlock) / threadsPerBlock, threadsPerBlock>>>(
            particles.gpu_position,
            particles.gpu_velocity,
            particles.radius,
            gpu_active_idxs,
            gpu_active_sizes,
            i,
            max_segs_cols,
            max_segs_rows,
            gpu_col_grid);
        hipDeviceSynchronize();
    }
    result["particles::collide::compare"] += stop.restart();

    blocks = (col_grid_size + threadsPerBlock - 1) / threadsPerBlock;
    // clearGrid<<<blocks, threadsPerBlock>>>(gpu_col_grid, col_grid_size);
    CUDA_CALL(hipMemset(gpu_active_sizes, 0, sizeof(uint32_t) * 4U));
    CUDA_CALL(hipMemset(gpu_col_grid, 0, sizeof(CompactVec) * col_grid_size));
    CUDA_CALL(hipMemset(gpu_active_idxs, 0, sizeof(uint32_t) * col_grid_size * 4U));
    result["particles::collide::reset"] += stop.restart();
    return result;
}
__global__ void constraintKernel(Particles& particles, vec2f area_min, vec2f area_max) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= max_particle_count) return;
    if(particles.gpu_position[i].x > area_max.x || particles.gpu_position[i].x < area_min.x)
        particles.gpu_velocity[i].x = 0;
    if(particles.gpu_position[i].y > area_max.y || particles.gpu_position[i].y < area_min.y)
        particles.gpu_velocity[i].y = 0;
    particles.gpu_position[i].x = fmaxf(particles.gpu_position[i].x, area_min.x);
    particles.gpu_position[i].y = fmaxf(particles.gpu_position[i].y, area_min.y);
    particles.gpu_position[i].x = fminf(particles.gpu_position[i].x, area_max.x);
    particles.gpu_position[i].y = fminf(particles.gpu_position[i].y, area_max.y);
}
void constraint(Particles& particles, AABB area) {
    int threadsPerBlock = 1024;
    int blocks = (max_particle_count + threadsPerBlock - 1) / threadsPerBlock;
    constraintKernel<<<blocks, threadsPerBlock>>>(particles, area.min, area.max);
    hipDeviceSynchronize();
    CUDA_KERNEL_CHECK();
}
ParticleSolveBlock::ParticleSolveBlock(Particles& p) : particles(p) {
    CUDA_CALL(hipMemcpy(particles.gpu_velocity, particles.velocity, sizeof(vec2f) * max_particle_count, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(particles.gpu_position, particles.position, sizeof(vec2f) * max_particle_count, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(particles.gpu_acceleration, particles.acceleration, sizeof(vec2f) * max_particle_count, hipMemcpyHostToDevice));
}
ParticleSolveBlock::~ParticleSolveBlock() {
    CUDA_CALL(hipMemcpy(particles.velocity, particles.gpu_velocity, sizeof(vec2f) * max_particle_count, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(particles.position, particles.gpu_position, sizeof(vec2f) * max_particle_count, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(particles.acceleration, particles.gpu_acceleration, sizeof(vec2f) * max_particle_count, hipMemcpyDeviceToHost));
}
void init(Particles& particles, AABB screen_area, float spacing, int seed) {
    auto w = screen_area.size().x - particles.radius * 2.f;
    int width = w / (particles.radius * 2 * spacing);
    CUDA_CALL(hipHostMalloc(&particles.position, sizeof(vec2f) * max_particle_count));
    CUDA_CALL(hipMalloc(&particles.gpu_position, sizeof(vec2f) * max_particle_count));
    CUDA_CALL(hipHostMalloc(&particles.acceleration, sizeof(vec2f) * max_particle_count));
    CUDA_CALL(hipMalloc(&particles.gpu_acceleration, sizeof(vec2f) * max_particle_count));
    CUDA_CALL(hipHostMalloc(&particles.velocity, sizeof(vec2f) * max_particle_count));
    CUDA_CALL(hipMalloc(&particles.gpu_velocity, sizeof(vec2f) * max_particle_count));

    for(int i = 0; i < max_particle_count; i++) { 
        particles.position[i].x = (i % width) * particles.radius * 2.f * spacing + screen_area.min.x;
        particles.position[i].y = (i / width) * particles.radius * 2.f * spacing + screen_area.min.y;

        particles.velocity[i] =  {0, 0};
        particles.acceleration[i] = {0, 0};
    }
}
void cleanup(Particles& particles) {
    CUDA_CALL(hipHostFree(particles.position));
    CUDA_CALL(hipFree(particles.gpu_position));
    CUDA_CALL(hipHostFree(particles.acceleration));
    CUDA_CALL(hipFree(particles.gpu_acceleration));
    CUDA_CALL(hipHostFree(particles.velocity));
    CUDA_CALL(hipFree(particles.gpu_velocity));
}

